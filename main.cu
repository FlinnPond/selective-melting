#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>
#include <string>
#include <vector>
#include <tuple>
//#include <hip/hip_runtime.h>
#include "vaporize.cuh"

__constant__ Data dat;

// __device__ inline double Max(double A, double B){if(A>B) return A; else return B;}
// __device__ inline double Min(double A, double B){if(A<B) return A; else return B;}

__global__ void CalcDiffusion(double* al, double* ti, double* al_next, double* ti_next, ST* state_field, double* heat) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int center = y*dat.Nx*dat.Nz + x*dat.Nz + z;
    printf("hey");
    if (x < dat.Nx && y<dat.Ny && z < dat.Nz && state_field[center] == FLUID) {
        int xp = y*dat.Nx*dat.Nz + (x+1)*dat.Nz + z;
        int xm = y*dat.Nx*dat.Nz + (x-1)*dat.Nz + z;
        int zp = y*dat.Nx*dat.Nz + x*dat.Nz + z + 1;
        int zm = y*dat.Nx*dat.Nz + x*dat.Nz + z - 1;
        int yp = (y+1)*dat.Nx*dat.Nz + x*dat.Nz + z;
        int ym = (y-1)*dat.Nx*dat.Nz + x*dat.Nz + z;
        double al_dole = (al[center]+ti[center] == 0) ? 0 : al[center] / (al[center] + ti[center]);
        double al_mass_loss = 0;
        double ti_mass_loss = 0;

        if (state_field[zm] == GAS) {
            al_mass_loss = Max(0.0,      al_dole  * activity_Al(heat[center], al_dole, dat.cts_Al_d) * mass_loss(heat[center], dat.cts_Al_d) * dat.tau);
            ti_mass_loss = Max(0.0, (1 - al_dole) * activity_Ti(heat[center], al_dole, dat.cts_Ti_d) * mass_loss(heat[center], dat.cts_Ti_d) * dat.tau);
        }
        double xmD = (state_field[xm] == FLUID) ? dat.D : 0;
        double xpD = (state_field[xp] == FLUID) ? dat.D : 0;
        double ymD = (state_field[ym] == FLUID) ? dat.D : 0;
        double ypD = (state_field[yp] == FLUID) ? dat.D : 0;
        double zmD = (state_field[zm] == FLUID) ? dat.D : 0;
        double zpD = (state_field[zp] == FLUID) ? dat.D : 0;

        al_next[center] = Max(0.0, al[center] + dat.tau * (
            (xmD*(al[xm] - al[center]) + xpD*(al[xp] - al[center]))/(dat.h*dat.h) + 
            (ymD*(al[ym] - al[center]) + ypD*(al[yp] - al[center]))/(dat.h*dat.h) + 
            (zmD*(al[zm] - al[center]) + zpD*(al[zp] - al[center]))/(dat.h*dat.h)
        ) - al_mass_loss / 0.027 / dat.h);

        ti_next[center] = Max(0.0, ti[center] + dat.tau * (
            (xmD*(ti[xm] - ti[center]) + xpD*(ti[xp] - ti[center]))/(dat.h*dat.h) + 
            (ymD*(ti[ym] - ti[center]) + ypD*(ti[yp] - ti[center]))/(dat.h*dat.h) + 
            (zmD*(ti[zm] - ti[center]) + zpD*(ti[zp] - ti[center]))/(dat.h*dat.h)
        ) - ti_mass_loss / 0.048 / dat.h);
        // if (x == 200 && z == 1 && y == 1) {
        //     printf("D: %e, al_next: %e, mass_loss: %e\n", dat.D, al_next[center], dat.tau * al_mass_loss / 0.048 / dat.h);
        // }

        // if (isnan(al_next[center])) {
        //     printf("X:%i, z: %i, al_next: %e, loss: %e, mass_loss/mu/h: %e, mass_loss: %e, activity_Al: %e, al_dole: %e \n",x, z, al_next[center], al_mass_loss / 0.027 / dat.hz, al_mass_loss, mass_loss(heat[center], dat.cts_Al_d) * dat.tau, activity_Al(heat[center], al_dole, dat.cts_Al_d), al_dole);
        //     al_next[center] = dat.al_0;
        // }
    }
}

__global__ void CalcHeatConduct(double *heat, double *heat_next, double *heat_source, double* al, double* ti, ST* state_field) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int center = y*dat.Nx*dat.Nz + x*dat.Nz + z;
    int xp = y*dat.Nx*dat.Nz + (x+1)*dat.Nz + z;
    int xm = y*dat.Nx*dat.Nz + (x-1)*dat.Nz + z;
    int zp = y*dat.Nx*dat.Nz + x*dat.Nz + z + 1;
    int zm = y*dat.Nx*dat.Nz + x*dat.Nz + z - 1;
    int yp = (y+1)*dat.Nx*dat.Nz + x*dat.Nz + z;
    int ym = (y-1)*dat.Nx*dat.Nz + x*dat.Nz + z;

    if (x < dat.Nx && y<dat.Ny && z < dat.Nz && (state_field[center] == FLUID || state_field[center] == SOLID)) {
        // double al_mass_frac = al[center] / (al[center] + ti[center] * dat.cts_Ti_d->MOL_MASS / dat.cts_Al_d->MOL_MASS);
        double heat_cond = (dat.cts_Al_d->HEAT_COND_0_SOL + dat.cts_Ti_d->HEAT_COND_0_SOL)/2;
        double heat_cap  = (dat.cts_Al_d->HEAT_CAP_SOL    + dat.cts_Ti_d->HEAT_CAP_SOL)/2;
        double density   = (dat.cts_Al_d->SOL_DENSITY     + dat.cts_Ti_d->SOL_DENSITY)/2;
        double D = heat_cond / (heat_cap * density);
        double topD = D;
        double energy_source = 0;

        if (state_field[zm] == GAS){
            energy_source = heat_source[y*dat.Nx + x] - heat_loss(heat[center], dat.cts_Al_d);
            energy_source /= (heat_cap * density);
            topD = 0;
        }

        heat_next[center] = heat[center] + dat.tau * (
            ((D)   *(heat[xm] - heat[center]) + (D)*(heat[xp] - heat[center]))/(dat.h*dat.h) +
            ((0)   *(heat[ym] - heat[center]) + (0)*(heat[yp] - heat[center]))/(dat.h*dat.h) +
            ((topD)*(heat[zm] - heat[center]) + (D)*(heat[zp] - heat[center]))/(dat.h*dat.h) + energy_source/dat.h
        );
        // if (x == 200 && z == 1 && y == 1) {
        //     printf("D: %e, heat_next: %e, source: %e\n", D, heat_next[center  ], dat.tau * energy_source/dat.hz);
        // }
    }
}

__global__ void UpdateState(ST* state_field, double* heat) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int center = y*dat.Nx*dat.Nz + x*dat.Nz + z;
    if(x<dat.Nx && y<dat.Ny && z<dat.Nz && (state_field[center] == SOLID || state_field[center] == FLUID)) {
        if     (heat[center] <  dat.cts_Ti_d->TEMP_LIQ) {state_field[center] = SOLID;}
        else if(heat[center] >= dat.cts_Ti_d->TEMP_LIQ) {state_field[center] = FLUID;}
    }
}

__global__ void CalcHeatSource(double *heat_source, int step) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int center = y*dat.Nx + x;
    if (x < dat.Nx && y < dat.Ny) {
        double current_time = step * dat.tau;
        double beam_xpos = fmax(0.0, current_time * dat.beam_vel - 2e-6) + dat.beam_xstart * dat.h;
        double beam_ypos = dat.beam_ystart*dat.h;
        double sigma = dat.beam_radius / 2;
        // heat_source[x] = 2 * dat.beam_power / (M_PI * dat.beam_radius * dat.beam_radius) * exp(-2 * pow(current_time * dat.beam_vel + dat.beam_start * dat.h - x * dat.h, 2) / pow(dat.beam_radius, 2));
        heat_source[center] = dat.beam_power / pow(sigma * sqrt(2*M_PI), 2) * exp(-0.5*pow((x*dat.h - beam_xpos) / sigma, 2) - 0.5*pow((y*dat.h - beam_ypos) / sigma, 2));
    }
}

__global__ void Move(double* al, double* ti, double* al_next, double* ti_next, ST* state_field){
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int z = blockIdx.y*blockDim.y + threadIdx.y;
    int center = x*dat.Nz+z;
    int right  = (x+1)*dat.Nz+z;
    if (state_field[center] == FLUID && x==dat.Nx-2) {al_next[center] = dat.al_0; ti_next[center] = dat.ti_0;}
    if (state_field[center] == FLUID) {al_next[center] = al[right]; ti_next[center] = ti[right];}
}

__host__ void sim_temp(int temp1, int temp2, int temp_step, double time_stop, int drop_rate, std::string drop_dir)
{
    for (int temp=temp1; temp<temp2; temp+=temp_step)
    {   
        Data dat_host;
        dat_host.init((double)temp, drop_rate);
        dat_host.time_stop = time_stop;
        
        int threadsPerBlock = 8;
        int blocksPerGridX = (dat_host.Nx + threadsPerBlock - 1) / threadsPerBlock;
        int blocksPerGridY = (dat_host.Ny + threadsPerBlock - 1) / threadsPerBlock;
        int blocksPerGridZ = (dat_host.Nz + threadsPerBlock - 1) / threadsPerBlock;
    
        dim3 blockShape(threadsPerBlock, threadsPerBlock, threadsPerBlock);
        dim3 gridShape(blocksPerGridX, blocksPerGridY, blocksPerGridZ);

        dim3 blockShapeSrc(threadsPerBlock, threadsPerBlock);
        dim3 gridShapeSrc(blocksPerGridX, blocksPerGridY);

        hipMemcpyToSymbol(HIP_SYMBOL(dat), &dat_host, sizeof(Data));
        std::cout << "copy data: " << hipGetErrorString(hipGetLastError()) << "\n";

        while (dat_host.tim < time_stop)
        {
            dat_host.step++;
            //std::cout << "temp " << temp << ", step " << dat_host.step << ": time = " << dat_host.tim << "; " << (dat_host.tim) / time_stop * 100 << "%" << std::endl;
            CalcHeatSource<<<gridShapeSrc, blockShapeSrc>>>(dat_host.heat_source_d, dat_host.step);
            hipDeviceSynchronize();

            CalcHeatConduct<<<gridShape, blockShape>>>(dat_host.heat_d, dat_host.heat_next_d, dat_host.heat_source_d, dat_host.al_d, dat_host.ti_d, dat_host.state_field_d);
            std::swap(dat_host.heat_next_d, dat_host.heat_d);
            hipDeviceSynchronize();

            UpdateState<<<gridShape, blockShape>>>(dat_host.state_field_d, dat_host.heat_d);
            hipDeviceSynchronize();
            
            CalcDiffusion<<<gridShape, blockShape>>>(dat_host.al_d, dat_host.ti_d, dat_host.al_next_d, dat_host.ti_next_d, dat_host.state_field_d, dat_host.heat_d);
            std::swap(dat_host.al_next_d, dat_host.al_d);
            std::swap(dat_host.ti_next_d, dat_host.ti_d);            
            if (dat_host.step % dat_host.drop_rate == 0) {
                hipDeviceSynchronize();
                dat_host.extract();
                std::ofstream output;
                output.open("data/dataT.txt");
                double maxT = 0;
                hipDeviceSynchronize();
                double scale = 1e6;
                for (int x = 0; x < dat_host.Nx; x++)
                {
                    for (int z = 0; z < dat_host.Nz; z++)
                    {
                        int center = 1*dat_host.Nx*dat_host.Nz + x*dat_host.Nz + z;
                        if (dat_host.heat_h[center] > maxT) {maxT = dat_host.heat_h[center];}
                        output << x*dat_host.h*scale << " " << z*dat_host.hz*scale << " " << dat_host.heat_h[center] << " " << dat_host.al_h[center]/dat_host.al_0 << " " << dat_host.ti_h[center]/dat_host.ti_0 << " " << dat_host.state_field_h[center] << "\n";
                    }
                    output << "\n";
                }
                output.close();
                std::cout << "Tmax is: " << maxT << "\n";
                float highest_temp = 6000;
                float melt_temp = dat_host.cts_Ti_h->TEMP_MELT;
                float melt_frac = melt_temp / highest_temp;

                Gnuplot gp;
                gp << "load \"scriptT.gp\" \n";
                std::cout << "set output \"" << drop_dir << "/Tmap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gp        << "set output \"" << drop_dir << "/Tmap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gp << "set xrange [" << (fmax(0.0, dat_host.step*dat_host.tau * dat_host.beam_vel - 2e-6) + (dat_host.beam_xstart - 100) * dat_host.h)*scale << ":" << (fmax(0.0, dat_host.step*dat_host.tau * dat_host.beam_vel - 2e-6) + (dat_host.beam_xstart + 100) * dat_host.h)*scale << "]; set yrange [" << 3*dat_host.Nz*dat_host.hz/2*scale*1/3 << ":" << -dat_host.Nz*dat_host.hz/2*scale*0.25  <<"]\n";
                // gp << "set xrange [0:" << dat_host.Nx * dat_host.h << "]; set yrange [" << 0 << ":* ]\n";
                gp << "set palette defined (0 \"black\"," << melt_frac/2 << "\"blue\", " << melt_frac << "\"red\", " << melt_frac << "\"#00efc1\", " << (1+melt_frac)/2 << "\"#ffff46\", 1 \"grey90\") \n";
                gp << "set cbrange [0:" << highest_temp << "] \n";
                gp << "set title \"Al-Ti melt pool heat map at t = " << std::setprecision(3) << dat_host.tim << ", T_{max} = " << (int)maxT << "\"; ";
                gp << "splot \"data/dataT.txt\" u 1:2:3 with pm3d\n";

                std::cout << "set output \"" << drop_dir << "/Tmapzoomed_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gp        << "set output \"" << drop_dir << "/Tmapzoomed_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gp << "set xrange [0:" << dat_host.Nx * dat_host.h*scale << "]; set yrange [" << 3*dat_host.Nz*dat_host.hz/2*scale << ":" << -dat_host.Nz*dat_host.hz/2*scale  <<"]\n";
                // gp << "set xrange [0:" << dat_host.Nx * dat_host.h << "]; set yrange [" << 0 << ":* ]\n";
                gp << "set palette defined (0 \"black\"," << melt_frac/2 << "\"blue\", " << melt_frac << "\"red\", " << melt_frac << "\"#00efc1\", " << (1+melt_frac)/2 << "\"#ffff46\", 1 \"grey90\") \n";
                gp << "set cbrange [0:" << highest_temp << "] \n";
                gp << "set title \"Al-Ti melt pool heat map at t = " << std::setprecision(3) << dat_host.tim << ", T_{max} = " << (int)maxT << "\"; ";
                gp << "splot \"data/dataT.txt\" u 1:2:3 with pm3d\n";


                Gnuplot gpc;
                gpc << "load \"scriptC.gp\" \n";
                std::cout << "set output \"" << drop_dir << "/cmap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gpc       << "set output \"" << drop_dir << "/cmap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n ";
                gpc << "set xrange [0:" << dat_host.Nx * dat_host.h*scale << "]; set yrange [" << 3*dat_host.Nz*dat_host.hz/2*scale << ":" << -dat_host.Nz*dat_host.hz/2*scale  <<"]\n";
                gpc << "set title \"Al concentration in Al-Ti melt pool at t = " << std::setprecision(3) << dat_host.tim << "\"; ";
                gpc << "splot \"data/dataT.txt\" u 1:2:4 with pm3d\n";
                
                std::cout << "set output \"" << drop_dir << "/cmapzoomed_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gpc       << "set output \"" << drop_dir << "/cmapzoomed_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n ";
                gpc << "set xrange [" << (fmax(0.0, dat_host.step*dat_host.tau * dat_host.beam_vel - 2e-6) + (dat_host.beam_xstart - 40) * dat_host.h)*scale << ":" << (fmax(0.0, dat_host.step*dat_host.tau * dat_host.beam_vel - 2e-6) + (dat_host.beam_xstart + 40) * dat_host.h)*scale << "]; set yrange [" << 3*dat_host.Nz*dat_host.hz/2*scale*1/12 << ":" << -dat_host.Nz*dat_host.hz/2*scale*0.12  <<"]\n";
                gpc << "set title \"Al concentration in Al-Ti melt pool at t = " << std::setprecision(3) << dat_host.tim << "\"; ";
                gpc << "splot \"data/dataT.txt\" u 1:2:4 with pm3d\n";


                Gnuplot gps;
                gpc << "load \"scriptState.gp\" \n";
                std::cout << "set output \"" << drop_dir << "/statemap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n";
                gpc       << "set output \"" << drop_dir << "/statemap_" << std::setfill('0') << std::setw(7) << (int)dat_host.step << ".png\"\n ";
                gpc << "set xrange [0:" << dat_host.Nx * dat_host.h*scale << "]; set yrange [" << 3*dat_host.Nz*dat_host.hz/2*scale << ":" << -dat_host.Nz*dat_host.hz/2*scale  <<"]\n";
                gpc << "set title \"State in Al-Ti melt pool at t = " << std::setprecision(3) << dat_host.tim << "\"; ";
                gpc << "splot \"data/dataT.txt\" u 1:2:6 with pm3d\n";
            }
            dat_host.tim += dat_host.tau;
            dat_host.deltaX += dat_host.beam_vel * dat_host.tau;
        }
        dat_host.clean();
    }
    hipDeviceSynchronize();
}

int main()
{
    // sim_temp(2700.0, 3300.0, 100.0, 1e-5, 1000, "plots/cmapT_anim11_45");
    // sim_temp(3300.0, 4400.0, 200.0, 1e-5, 1000, "plots/cmapT_anim11_45");
    sim_temp(3000.0, 3001.0, 200.0, 2e-3, 1000, "plots/maps_anim11_05");

    return 0;
}
